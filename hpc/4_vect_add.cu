#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>

using namespace std;

// CUDA kernel for vector addition
__global__ void vectorAdd(int* a, int* b, int* c, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < size) {
        c[tid] = a[tid] + b[tid];
    }
}

int main() {
    int size;
    cout << "Enter the size of the vectors: ";
    cin >> size;

    vector<int> a(size), b(size), c(size);

    cout << "Enter elements of vector A:\n";
    for (int i = 0; i < size; i++) {
        cin >> a[i];
    }

    cout << "Enter elements of vector B:\n";
    for (int i = 0; i < size; i++) {
        cin >> b[i];
    }

    int* dev_a, * dev_b, * dev_c;
    hipMalloc((void**)&dev_a, size * sizeof(int));
    hipMalloc((void**)&dev_b, size * sizeof(int));
    hipMalloc((void**)&dev_c, size * sizeof(int));

    hipMemcpy(dev_a, a.data(), size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b.data(), size * sizeof(int), hipMemcpyHostToDevice);

    int blockSize = 128;
    int gridSize = (size + blockSize - 1) / blockSize;
    vectorAdd<<<gridSize, blockSize>>>(dev_a, dev_b, dev_c, size);
    hipDeviceSynchronize();

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        cout << "CUDA error: " << hipGetErrorString(err) << endl;
        return -1;
    }

    hipMemcpy(c.data(), dev_c, size * sizeof(int), hipMemcpyDeviceToHost);

    cout << "Result of vector addition:\n";
    for (int i = 0; i < size; i++) {
        cout << a[i] << " + " << b[i] << " = " << c[i] << endl;
    }

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}